#include <torch/extension.h>
#include "getcuda.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
using namespace std;

int get_cuda(){
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    cout << "GPU device " << dev << ": " << devProp.name << endl;
    cout << "Number of SM:" << devProp.multiProcessorCount << endl;
    cout << "max_threads_per_block:" << devProp.maxThreadsPerBlock << endl;
    cout << "max_shared_memory_per_block:" << devProp.sharedMemPerBlock << " Bytes" << endl;
    return 0;
}