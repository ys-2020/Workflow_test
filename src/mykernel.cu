#include <iostream>
#include <hip/hip_runtime.h>


void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}


__global__ void add_kernel(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}


extern "C"{
    float my_test() {
        
        int n = 4096;

        float *h_A, *h_B, *h_C;
        hipHostMalloc(&h_A, n * sizeof(float), hipHostMallocDefault);
        hipHostMalloc(&h_B, n * sizeof(float), hipHostMallocDefault);
        hipHostMalloc(&h_C, n * sizeof(float), hipHostMallocDefault);
        random_init(h_A, n);
        random_init(h_B, n);

        float *d_A, *d_B, *d_C;
        hipMalloc(&d_A, n * sizeof(float));
        hipMalloc(&d_B, n * sizeof(float));
        hipMalloc(&d_C, n * sizeof(float));

        hipMemcpy(d_A, h_A, n * sizeof(float), hipMemcpyDefault);
        hipMemcpy(d_B, h_B, n * sizeof(float), hipMemcpyDefault);

        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        hipEventRecord(start);
        add_kernel<<<((n + 255) / 256), 256>>>(d_A, d_B, d_C, n);
        hipEventRecord(end);
        hipEventSynchronize(end);

        hipMemcpy(h_C, d_C, n * sizeof(float), hipMemcpyDefault);

        printf("%f\n",h_C[0]);
        float ret = h_C[0];

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipHostFree(h_A);
        hipHostFree(h_B);
        hipHostFree(h_C);
        return ret;
    }
}