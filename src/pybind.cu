#include <pybind11/pybind11.h>
#include <torch/extension.h>
#include <iostream>
#include <dlfcn.h>

namespace py = pybind11;

extern "C" {
float call_my_kernel() {
    // hipSetDevice(0);

// A better way to ref? Relative reference can lead to errors when the python file test.py is moved.
    void* handle = dlopen("./src/my_lib.so", RTLD_LAZY);
    if (!handle) {
        throw std::runtime_error("[pybind.cu] Failed to load shared object file. Please check the path to *.so.");
    }
    // Get a pointer to the CUDA kernel
    typedef float (*my_kernel_func)();
    my_kernel_func my_kernel = reinterpret_cast<my_kernel_func>(dlsym(handle, "my_test"));
    if (!my_kernel) {
        throw std::runtime_error("[pybind.cu] Failed to get pointer to CUDA kernel.");
    }

    float a = my_kernel();

    dlclose(handle);
    // hipDeviceReset();
    return a;
}
}
// Define the PyBind11 module
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("call_my_kernel", &call_my_kernel, "Call the CUDA kernel in mylib.so");
}
